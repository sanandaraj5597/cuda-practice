
#include <hip/hip_runtime.h>
#include<mma.h>
#include<stdio.h>
using namespace nvcuda;

#define CUDA_CHECK_RETURN(X) X
#define NUM_ITERS 30

// Define some error checking macros.
#define cudaErrCheck(stat)                         \
    {                                              \
        cudaErrCheck_((stat), __FILE__, __LINE__); \
    }

void cudaErrCheck_(hipError_t stat, const char *file, int line)
{
    if (stat != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

const int M = 327660;
const int N = 1536;
const int K = 512;

const int num_threads = 1024;
const int smem = K * 32;

__global__ void wmma_kernel(half* a, half* b, float* c){

   __shared__ half SMEM[smem];
   int warp_id = threadIdx.x/32;
   int work_per_warp = N/(WMMA_N*32);

   wmma::fragment<wmma::matrix_a,WMMA_M,WMMA_N,WMMA_K,half,wmma::row_major> frag_a;
   wmma::fragment<wmma::matrix_b,WMMA_M,WMMA_N,WMMA_K,half,wmma::row_major> frag_b;
   wmma::fragment<wmma::accumulator,WMMA_M,WMMA_N,WMMA_K,float> frag_c;

   for(int it=0; it<NUM_ITERS; it++){
    for(int m=0; m<(M/16); m++){
     for(int i=0 ; i<((K*16)/num_threads); i++)
      SMEM[threadIdx.x + (i * num_threads)] = a[threadIdx.x * (i * num_threads)];
 
     __syncthreads();
  
     for(int i=0 ; i<1; i++){
      for(int j=0; j<work_per_warp; j++){
       wmma::fill_fragment(frag_c,0.0f);
       for(int k=0; k<(K/WMMA_K); k++){
        wmma::load_matrix_sync(frag_a,&SMEM[(i*K*WMMA_M) + (k*WMMA_K)],K);
        wmma::load_matrix_sync(frag_b,&b[(j*WMMA_N) + work_per_warp*warp_id*WMMA_N + (k*WMMA_K*N)],N);
  
        wmma::mma_sync(frag_c,frag_a,frag_b,frag_c);
       }
      wmma::store_matrix_sync(&c[(i*WMMA_M*N)+(m*16*N)+((j+(warp_id*work_per_warp))*WMMA_N)],frag_c,N,wmma::mem_row_major);
      }
     }
    }
   }
}

int main(){

 half *d_a, *h_a, *d_b, *h_b;
 float *d_c, *h_c;
 h_c = new float[M*N];
 h_b = new half[K*N];
 h_a = new half[M*K];
 hipMalloc(&d_a, M*K*sizeof(half));
 hipMalloc(&d_b, K*N*sizeof(half));
 hipMalloc(&d_c, M*N*sizeof(float));
 for (int i = 0; i < M*K; i++)
   h_a[i] = 1.0f;
 for (int i = 0; i < N*K; i++)
   h_b[i] = 1.0f;
 hipMemcpy(d_a, h_a, M*K*sizeof(half), hipMemcpyHostToDevice);
 hipMemcpy(d_b, h_b, K*N*sizeof(half), hipMemcpyHostToDevice);
 
 hipEvent_t start, stop;
 CUDA_CHECK_RETURN(hipEventCreate(&start));
 CUDA_CHECK_RETURN(hipEventCreate(&stop));

 CUDA_CHECK_RETURN(hipEventRecord(start));
 wmma_kernel<<<1,num_threads>>>(d_a, d_b, d_c);
 cudaErrCheck(hipGetLastError());
 CUDA_CHECK_RETURN(hipEventRecord(stop));

 hipMemcpy(h_c, d_c, M*N*sizeof(float), hipMemcpyDeviceToHost);

 float elapsedTime;
 hipEventElapsedTime(&elapsedTime, start, stop);

 printf("Elapsed Time : %f\n",elapsedTime);

 return 0;
}
